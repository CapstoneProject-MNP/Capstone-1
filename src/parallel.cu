#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
using namespace std;
# define INF 0x3f3f3f3f
void printGrpah( int *a, int *e, int *b, int *c, int *d, int num_v, int num_e ) 
{
    for(int v = 0; v<num_v; v++)
    {
        if(a[v] != -1)
        {
           int start = a[v];
            int end = e[v];
            printf("%d-->", v);
            while( start <= end )
            {
                printf("[%d, {%d, %d} ]---", b[start], c[start], d[start]);
                start++;
            }
        }
        printf("\n");
    }
    return;
}

/************************Host Dijkstras Algorithm for Intitial Seed Path Start ***********************************/

vector<pair<int, pair<int,int>>> seedPathDijkstra(int src, int dest, int* h_offset, int* h_offend, int* h_earr, int* h_carr, int* h_sarr, int num_v, int num_e) 
{ 
	priority_queue< pair<int, int>, vector <pair<int, int>> , greater <pair<int, int>> > pq;
    int V = num_v;
	vector<int> dist(V, INF);
    vector<pair<int, pair<int,int>>> parent(V, {-1, {-1,-1}});
    vector<pair<int, pair<int,int>>> seedPath;

	pq.push(make_pair(0, src));
	dist[src] = 0;
	
	vector<bool> f(V, false);

	while (!pq.empty())
	{
		int u = pq.top().second;
		pq.pop();
		f[u] = true;

        int index = h_offset[u];
        int end = h_offend[u];

		for (int i=index; i <=end; i++)
		{
            int v = h_earr[i];
            int cost = h_carr[i];
            int score = h_sarr[i];

			// If there is shorted path to v through u.
			if (f[v] == false && dist[v] > dist[u] + cost)
			{
				// Updating distance of v
				dist[v] = dist[u] + cost;
				pq.push(make_pair(dist[v], v));
                pair<int, pair<int,int>> temp;
                temp.first = u;
                temp.second.first = cost;
                temp.second.second = score;
                parent[v] = temp;
			}
		}
	}

    //Find shortest path from src to some dest
    stack<pair<int, pair<int,int>>> path;
    int nd = dest;
    while(parent[nd].first != -1)
    {
        path.push(parent[nd]);
        nd = parent[nd].first;
    }

    while(!path.empty())
    {
        seedPath.push_back(path.top());
        path.pop();
    }
    seedPath.push_back({dest, {0,0}});

    int sz_p = seedPath.size();
    for(int i=sz_p-1; i>0; i--)
    {
        seedPath[i].second.first = seedPath[i-1].second.first;
        seedPath[i].second.second = seedPath[i-1].second.second;        
    }
    seedPath[0].second.first = 0;
    seedPath[0].second.second = 0;

    for(int i=1; i<sz_p; i++)
    {
        seedPath[i].second.first += seedPath[i-1].second.first;
        seedPath[i].second.second += seedPath[i-1].second.second;        
    }
        

    return seedPath;      
} 
/********************* Dijkstra seed PAth end*******************************************/

/**************************** Kernel code*********/

__device__ int minDistance(int* dist,int* sptSet, int num_v) 
{ 
    int min = 100000000, min_index; 
    for (int v = 0; v < num_v; v++) 
        if (sptSet[v] == 0 && dist[v] <= min) 
        {    min = dist[v];
            min_index = v;
        } 
  
    return min_index; 
} 

__device__ int EucliDist(int src, int dest, int* dev_offset, int* dev_offend, int* dev_earr, int* dev_carr, int num_v, int num_e) 
{
    int dist[11];
    int sptSet[11];
 
    for(int i=0; i<num_v; i++)
    {
        dist[i] = 100000000;
        sptSet[i] = 0;
    }

    dist[src] = 0; 
  
    for (int count = 0; count < num_v - 1; count++) { 

        int u = minDistance(dist, sptSet, num_v); 
        sptSet[u] = 1; 
        int index = dev_offset[u];
        int end = dev_offend[u];
  
        // Update dist value of the adjacent vertices of the picked vertex. 
        for (int i = index; i <= end; i++) 
        {    
            int v = dev_earr[i]; //adjacent vertex of u
            int cost = dev_carr[i];
            if (!sptSet[v] && dist[u] != 100000000 
                && dist[u] + cost < dist[v]) 
                dist[v] = dist[u] + cost; 
        }
    } 
    return dist[dest];
} 
__device__ struct BestVertex{
    int vertex;
    float gamma;
    int score;
    int cost;
};

__device__ float GammaValue(int cost, int score, int De)
{
    float g = float(1 + score)/float(cost + De);   
    return g;
}

/* Find Best Successor */
__device__ struct BestVertex BestSuccessor(int* dev_offset, int* dev_offend,int* dev_earr,int* dev_carr,int* dev_sarr,int num_v,int num_e, int front_tail, int back_tail )
{
    struct BestVertex VGCS;   // best successor struct
    float gamma = -1;
 
    int start_index = dev_offset[front_tail] ;
    int end_index = dev_offend[front_tail];
	for (int i=start_index; i <= end_index; i++)  //for all adjacent outgoing edge
	{
        int v = dev_earr[i];
        int cost = dev_carr[i];
        int score = dev_sarr[i];
  
        int De = EucliDist(v, back_tail, dev_offset,dev_offend, dev_earr, dev_carr, num_v, num_e);
        //int De = 1;
        float g = GammaValue(cost, score, De);
        
        if( g > gamma )
        {
            gamma = g;
            VGCS.vertex = v;
            VGCS.gamma = g;
            VGCS.score = score;
            VGCS.cost = cost;
        }
    }
    return VGCS;
}

/* Find Best Successor */
__device__ struct BestVertex BestPredecessor(int* dev_offset,int* dev_offend,int* dev_earr,int* dev_carr,int* dev_Ioffset, int* dev_Ioffend,int* dev_Iearr,int* dev_Icarr,int* dev_Isarr, int num_v,int num_e, int front_tail, int back_tail )
{
    struct BestVertex VGCS;   // best successor struct
    float gamma = -1;
 
    int start_index = dev_Ioffset[back_tail] ;
    int end_index = dev_Ioffend[back_tail];
	for (int i=start_index; i <= end_index; i++)  //for all adjacent incoming edge
	{
        int v = dev_Iearr[i];
        int cost = dev_Icarr[i];
        int score = dev_Isarr[i];
  
        int De = EucliDist(front_tail, v, dev_offset,dev_offend, dev_earr, dev_carr, num_v, num_e);
        //int De = 1;
        float g = GammaValue(cost, score, De);
        
        if( g > gamma )
        {
            gamma = g;
            VGCS.vertex = v;
            VGCS.gamma = g;
            VGCS.score = score;
            VGCS.cost = cost;
        }
    }
    return VGCS;
}

__global__ void display(int pathSize, int* d_seedarr,int* dev_offset, int* dev_offend,int* dev_earr,int* dev_carr,int* dev_sarr,
                        int num_v,int num_e,int* dev_Ioffset, int* dev_Ioffend,int* dev_Iearr,int* dev_Icarr,int* dev_Isarr) 
{
    int i = EucliDist(0, 10, dev_offset, dev_offend, dev_earr, dev_carr, num_v, num_e);
    float gamma = GammaValue(2,1,2);
    struct BestVertex Succ = BestSuccessor(dev_offset,dev_offend,dev_earr,dev_carr,dev_sarr,num_v,num_e, 1, 5);
    struct BestVertex Pred = BestPredecessor(dev_offset,dev_offend,dev_earr,dev_carr,dev_Ioffset,dev_Ioffend,dev_Iearr,dev_Icarr,dev_Isarr,num_v,num_e, 1, 5);
    printf("-----%d-----%d--------%f-------%d\n", i, Succ.score, gamma, Succ.vertex);
    printf("-Pred--sc--%d---v--%d------gm--%f-----cost--%d", Pred.score, Pred.vertex,Pred.gamma,Pred.cost);
    return;
}
/****************************************************/
int main()
{
/****************************Graph Initialisation in host and device
*************************************************************************************/  
    cout<<"hello";
    int num_v = 11;
    int num_e = 10;   
//Graph for storing outgoing edges
    int h_offset[num_v] = {0,1,4,5,6,7,-1,8,9,-1,-1};
    int h_offend[num_v] = {0,3,4,5,6,7,-1,8,9,-1,-1};     
    int h_earr[num_e] = {1,2,7,3,5,4,5,10,8,5};
    int h_carr[num_e] = {2,5,6,16,7,17,4,3,9,3}; 
    int h_sarr[num_e] = {1,3,4,10,0,15,2,5,15,0};    //variables declared for representing graph in cpu
    int *dev_offset, *dev_offend, *dev_earr, *dev_carr, *dev_sarr;  
//Graph for storing incoming edges
    int h_Ioffset[num_v] = {-1,0,1,2,3,4,-1,7,8,-1,9};
    int h_Ioffend[num_v] = {-1,0,1,2,3,6,-1,7,8,-1,9};     
    int h_Iearr[num_e] = {0,1,1,3,4,2,8,1,7,5};
    int h_Icarr[num_e] = {2,5,16,17,4,7,3,6,9,3}; 
    int h_Isarr[num_e] = {1,3,10,15,2,0,0,4,15,5};    //variables declared for representing graph in cpu
    int *dev_Ioffset, *dev_Ioffend, *dev_Iearr, *dev_Icarr, *dev_Isarr;  

    hipMalloc( (void**)&dev_offset, num_v*sizeof(int) ) ;
    hipMalloc( (void**)&dev_offend, num_v*sizeof(int) ) ;    
    hipMalloc( (void**)&dev_earr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_carr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_sarr, num_e*sizeof(int) ) ; 
    hipMalloc( (void**)&dev_Ioffset, num_v*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Ioffend, num_v*sizeof(int) ) ;    
    hipMalloc( (void**)&dev_Iearr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Icarr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Isarr, num_e*sizeof(int) ) ; 
    hipDeviceSynchronize();
    hipMemcpy( dev_offset, h_offset, num_v*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_offend, h_offend, num_v*sizeof(int), hipMemcpyHostToDevice ) ;    
    hipMemcpy( dev_earr, h_earr, num_e*sizeof(int), hipMemcpyHostToDevice ) ; 
    hipMemcpy( dev_carr, h_carr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_sarr, h_sarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Ioffset, h_Ioffset, num_v*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Ioffend, h_Ioffend, num_v*sizeof(int), hipMemcpyHostToDevice ) ;    
    hipMemcpy( dev_Iearr, h_Iearr, num_e*sizeof(int), hipMemcpyHostToDevice ) ; 
    hipMemcpy( dev_Icarr, h_Icarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Isarr, h_Isarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipDeviceSynchronize();

/****************************Seed Path find, copy to device
*************************************************************************************/    

    //find intial sead path
    vector<pair<int, pair<int,int>>> seedPath;
    seedPath = seedPathDijkstra( 0, 10, h_offset, h_offend,h_earr, h_carr, h_sarr, num_v, num_e);
    
    int pathSize = seedPath.size();
    int h_seedarr[3][pathSize];    
    
    cout<<"seedPath---";
    int iseed=0;
    for(auto v : seedPath)
    {
        cout<<"("<<v.first<<", "<<"("<<v.second.first<<","<<v.second.second<<")"<<endl;
        h_seedarr[0][iseed] = v.first;
        h_seedarr[1][iseed] = v.second.first;
        h_seedarr[2][iseed] = v.second.second;
        iseed++;
    }
 
 //alloate memory to device and copy
    int *d_seedarr;
    hipMalloc((void **)&d_seedarr,3*pathSize*sizeof(int));
    hipMemcpy( d_seedarr, h_seedarr, 3*pathSize*sizeof(int), hipMemcpyHostToDevice ) ;
    hipDeviceSynchronize();

/******************************************************************************************************/
 //Kernel call to check functionalities
  display<<<1,1>>>( pathSize, d_seedarr, dev_offset, dev_offend,dev_earr, dev_carr, dev_sarr, num_v, num_e, dev_Ioffset, dev_Ioffend,dev_Iearr, dev_Icarr, dev_Isarr );
    hipDeviceSynchronize();
/*****************************************************************************************************/ 
    hipFree(dev_offset);
    hipFree(dev_offend); 
    hipFree(dev_earr);
    hipFree(dev_carr);
    hipFree(dev_sarr);
    hipFree(d_seedarr); 
    hipFree(dev_Ioffset);
    hipFree(dev_Ioffend); 
    hipFree(dev_Iearr);
    hipFree(dev_Icarr);
    hipFree(dev_Isarr);
    hipDeviceSynchronize();
    return 0;

}