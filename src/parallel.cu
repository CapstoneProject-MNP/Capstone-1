#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
using namespace std;
# define INF 0x3f3f3f3f

vector<vector<pair<int, pair<int,int>>>> graph;
vector<vector<pair<int, pair<int,int>>>> incoming_edges;

/************************Host Dijkstras Algorithm for Intitial Seed Path Start ***********************************/

vector<pair<int, pair<int,int>>> seedPathDijkstra(int src, int dest, int* h_offset, int* h_offend, int* h_earr, int* h_carr, int* h_sarr, int num_v, int num_e) 
{ 
	priority_queue< pair<int, int>, vector <pair<int, int>> , greater <pair<int, int>> > pq;
    int V = num_v;
	vector<int> dist(V, INF);
    vector<pair<int, pair<int,int>>> parent(V, {-1, {-1,-1}});
    vector<pair<int, pair<int,int>>> seedPath;

	pq.push(make_pair(0, src));
	dist[src] = 0;
	
	vector<bool> f(V, false);

	while (!pq.empty())
	{
		int u = pq.top().second;
		pq.pop();
		f[u] = true;

        int index = h_offset[u];
        int end = h_offend[u];

		for (int i=index; i <=end; i++)
		{
            int v = h_earr[i];
            int cost = h_carr[i];
            int score = h_sarr[i];

			// If there is shorted path to v through u.
			if (f[v] == false && dist[v] > dist[u] + cost)
			{
				// Updating distance of v
				dist[v] = dist[u] + cost;
				pq.push(make_pair(dist[v], v));
                pair<int, pair<int,int>> temp;
                temp.first = u;
                temp.second.first = cost;
                temp.second.second = score;
                parent[v] = temp;
			}
		}
	}

    //Find shortest path from src to some dest
    stack<pair<int, pair<int,int>>> path;
    int nd = dest;
    while(parent[nd].first != -1)
    {
        path.push(parent[nd]);
        nd = parent[nd].first;
    }

    while(!path.empty())
    {
        seedPath.push_back(path.top());
        path.pop();
    }
    seedPath.push_back({dest, {0,0}});

    int sz_p = seedPath.size();
    for(int i=sz_p-1; i>0; i--)
    {
        seedPath[i].second.first = seedPath[i-1].second.first;
        seedPath[i].second.second = seedPath[i-1].second.second;        
    }
    seedPath[0].second.first = 0;
    seedPath[0].second.second = 0;

    for(int i=1; i<sz_p; i++)
    {
        seedPath[i].second.first += seedPath[i-1].second.first;
        seedPath[i].second.second += seedPath[i-1].second.second;        
    }
        

    return seedPath;      
} 
/********************* Dijkstra seed PAth end*******************************************/

/**************************** Kernel code*********/

__device__ int minDistance(int* dist,int* sptSet, int num_v) 
{ 
    int min = 100000000, min_index; 
    for (int v = 0; v < num_v; v++) 
        if (sptSet[v] == 0 && dist[v] <= min) 
        {    min = dist[v];
            min_index = v;
        } 
  
    return min_index; 
} 

__device__ int EucliDist(int src, int dest, int* dev_offset, int* dev_offend, int* dev_earr, int* dev_carr, int num_v, int num_e) 
{
    int dist[11];
    int sptSet[11];
 
    for(int i=0; i<num_v; i++)
    {
        dist[i] = 100000000;
        sptSet[i] = 0;
    }

    dist[src] = 0; 
  
    for (int count = 0; count < num_v - 1; count++) { 

        int u = minDistance(dist, sptSet, num_v); 
        sptSet[u] = 1; 
        int index = dev_offset[u];
        int end = dev_offend[u];
  
        // Update dist value of the adjacent vertices of the picked vertex. 
        for (int i = index; i <= end; i++) 
        {    
            int v = dev_earr[i]; //adjacent vertex of u
            int cost = dev_carr[i];
            if (!sptSet[v] && dist[u] != 100000000 
                && dist[u] + cost < dist[v]) 
                dist[v] = dist[u] + cost; 
        }
    } 
    return dist[dest];
} 
__device__ struct BestVertex{
    int vertex;
    float gamma;
    int score;
    int cost;
};

__device__ float GammaValue(int cost, int score, int De)
{
    float g = float(1 + score)/float(cost + De);   
    return g;
}

/* Find Best Successor */
__device__ struct BestVertex BestSuccessor(int* dev_offset, int* dev_offend,int* dev_earr,int* dev_carr,int* dev_sarr,int num_v,int num_e, int front_tail, int back_tail )
{
    struct BestVertex VGCS;   // best successor struct
    float gamma = -1;
 
    int start_index = dev_offset[front_tail] ;
    int end_index = dev_offend[front_tail];
	for (int i=start_index; i <= end_index; i++)  //for all adjacent outgoing edge
	{
        int v = dev_earr[i];
        int cost = dev_carr[i];
        int score = dev_sarr[i];
  
        int De = EucliDist(v, back_tail, dev_offset,dev_offend, dev_earr, dev_carr, num_v, num_e);
        //int De = 1;
        float g = GammaValue(cost, score, De);
        
        if( g > gamma )
        {
            gamma = g;
            VGCS.vertex = v;
            VGCS.gamma = g;
            VGCS.score = score;
            VGCS.cost = cost;
        }
    }
    return VGCS;
}

/* Find Best Successor */
__device__ struct BestVertex BestPredecessor(int* dev_offset,int* dev_offend,int* dev_earr,int* dev_carr,int* dev_Ioffset, int* dev_Ioffend,int* dev_Iearr,int* dev_Icarr,int* dev_Isarr, int num_v,int num_e, int front_tail, int back_tail )
{
    struct BestVertex VGCS;   // best successor struct
    float gamma = -1;
 
    int start_index = dev_Ioffset[back_tail] ;
    int end_index = dev_Ioffend[back_tail];
	for (int i=start_index; i <= end_index; i++)  //for all adjacent incoming edge
	{
        int v = dev_Iearr[i];
        int cost = dev_Icarr[i];
        int score = dev_Isarr[i];
  
        int De = EucliDist(front_tail, v, dev_offset,dev_offend, dev_earr, dev_carr, num_v, num_e);
        //int De = 1;
        float g = GammaValue(cost, score, De);
        
        if( g > gamma )
        {
            gamma = g;
            VGCS.vertex = v;
            VGCS.gamma = g;
            VGCS.score = score;
            VGCS.cost = cost;
        }
    }
    return VGCS;
}

__global__ void display(int pathSize, int* d_seedarr,int* dev_offset, int* dev_offend,int* dev_earr,int* dev_carr,int* dev_sarr,
                        int num_v,int num_e,int* dev_Ioffset, int* dev_Ioffend,int* dev_Iearr,int* dev_Icarr,int* dev_Isarr) 
{
    int i = EucliDist(0, 10, dev_offset, dev_offend, dev_earr, dev_carr, num_v, num_e);
    float gamma = GammaValue(2,1,2);
    struct BestVertex Succ = BestSuccessor(dev_offset,dev_offend,dev_earr,dev_carr,dev_sarr,num_v,num_e, 1, 5);
    struct BestVertex Pred = BestPredecessor(dev_offset,dev_offend,dev_earr,dev_carr,dev_Ioffset,dev_Ioffend,dev_Iearr,dev_Icarr,dev_Isarr,num_v,num_e, 1, 5);
    printf("-----%d-----%d--------%f-------%d\n", i, Succ.score, gamma, Succ.vertex);
    printf("-Pred--sc--%d---v--%d------gm--%f-----cost--%d", Pred.score, Pred.vertex,Pred.gamma,Pred.cost);
    return;
}
/********************************************Device code end
*************************************************************************************************/

void DataPreprocessing(int count_nodes, string edge_file)
{
	string line;
    ifstream edgeFile;
    edgeFile.open(edge_file);
 
    //Build Graph
    graph.resize(count_nodes);
    incoming_edges.resize(count_nodes);
    while(edgeFile>>line)
    {
        //split the line by the delimiter ','
        vector<string>v;
        stringstream ss(line);
        while(ss.good())
        {
            string substr;
            getline(ss, substr, ',');
            v.push_back(substr);
        }

        //Extract the nodes, score and cost
        
        graph[stoi(v[0])].push_back(make_pair(stoi(v[1]),make_pair(stoi(v[2]),stoi(v[3]))));

        //store the incoming edge of a node
        incoming_edges[stoi(v[1])].push_back(make_pair(stoi(v[0]),make_pair(stoi(v[2]),stoi(v[3]))));
    }
}

void printGrpah( int *off, int *end, int *e, int *c, int *s, int num_v, int num_e ) 
{
    for(int v = 0; v<num_v; v++)
    {
        if(off[v] != -1)
        {
           int start = off[v];
            int en = end[v];
            printf("%d-->", v);
            while( start <= en )
            {
                printf("[%d, {%d, %d} ]---", e[start], c[start], s[start]);
                start++;
            }
        }
        printf("\n");
    }
    return;
}

void outAdjtoCSR(int* h_offset, int* h_offend, int* h_earr, int* h_carr, int* h_sarr)
{
    int ie = 0;
    for(int i=0; i<graph.size(); i++)
    {
        h_offset[i] = ( graph[i].size() == 0 )? -1 : ie;
        for(int j=0; j<graph[i].size(); j++)
        {
        	pair<int, pair<int,int>> node = graph[i][j];
            h_earr[ie] = node.first;
            h_carr[ie] = node.second.first;
            h_sarr[ie] = node.second.second;
            ie++;
        }
        h_offend[i] = ( graph[i].size() == 0 )? -1 : ie-1;
        cout<<endl;
    }
}

void inAdjtoCSR(int* h_Ioffset, int* h_Ioffend, int* h_Iearr, int* h_Icarr, int* h_Isarr)
{
    int ie = 0;
    for(int i=0; i<incoming_edges.size(); i++)
    {
        h_Ioffset[i] = ( incoming_edges[i].size() == 0 )? -1 : ie;
        for(int j=0; j<incoming_edges[i].size(); j++)
        {
        	pair<int, pair<int,int>> node = incoming_edges[i][j];
            h_Iearr[ie] = node.first;
            h_Icarr[ie] = node.second.first;
            h_Isarr[ie] = node.second.second;
            ie++;
        }
        h_Ioffend[i] = ( incoming_edges[i].size() == 0 )? -1 : ie-1;
        cout<<endl;
    }
}


int main()
{
/****************************Graph Initialisation in host 
*************************************************************************************/ 
    int num_v = 11;
    int num_e = 9;

    string edge_file;
    edge_file = "sample.txt";

    //Graph for storing outgoing edges
    int h_offset[num_v], h_offend[num_v], h_earr[num_e], h_carr[num_e], h_sarr[num_e]; 
 
    //Graph for storing incoming edges
    int h_Ioffset[num_v], h_Ioffend[num_v], h_Iearr[num_e], h_Icarr[num_e], h_Isarr[num_e];

    DataPreprocessing(num_v, edge_file);

    cout<<"outgoing adjacency graph"<<endl;
    for(auto n:graph)
    {
         for(auto v:n)
         cout<<"("<<v.first<<", "<<"("<<v.second.first<<","<<v.second.second<<")";
         cout<<endl;
    }
    cout<<"incoming adjacency graph"<<endl;
    for(auto n:incoming_edges)
    {
         for(auto v:n)
         cout<<"("<<v.first<<", "<<"("<<v.second.first<<","<<v.second.second<<")";
         cout<<endl;
    }

    //adjacency to csr 
    outAdjtoCSR(h_offset, h_offend, h_earr, h_carr, h_sarr);
    inAdjtoCSR(h_Ioffset, h_Ioffend, h_Iearr, h_Icarr, h_Isarr);
    cout<<"print csr graphs"<<endl;
    printGrpah( h_offset, h_offend, h_earr, h_carr, h_sarr, num_v, num_e ) ;
    printGrpah( h_Ioffset, h_Ioffend, h_Iearr, h_Icarr, h_Isarr, num_v, num_e ) ;

/****************************CSR Graph Initialisation in device 
****************************************************************************************************/ 

    int *dev_Ioffset, *dev_Ioffend, *dev_Iearr, *dev_Icarr, *dev_Isarr;  
    int *dev_offset, *dev_offend, *dev_earr, *dev_carr, *dev_sarr;

    hipMalloc( (void**)&dev_offset, num_v*sizeof(int) ) ;
    hipMalloc( (void**)&dev_offend, num_v*sizeof(int) ) ;    
    hipMalloc( (void**)&dev_earr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_carr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_sarr, num_e*sizeof(int) ) ; 
    hipMalloc( (void**)&dev_Ioffset, num_v*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Ioffend, num_v*sizeof(int) ) ;    
    hipMalloc( (void**)&dev_Iearr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Icarr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_Isarr, num_e*sizeof(int) ) ; 
    hipDeviceSynchronize();
    hipMemcpy( dev_offset, h_offset, num_v*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_offend, h_offend, num_v*sizeof(int), hipMemcpyHostToDevice ) ;    
    hipMemcpy( dev_earr, h_earr, num_e*sizeof(int), hipMemcpyHostToDevice ) ; 
    hipMemcpy( dev_carr, h_carr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_sarr, h_sarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Ioffset, h_Ioffset, num_v*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Ioffend, h_Ioffend, num_v*sizeof(int), hipMemcpyHostToDevice ) ;    
    hipMemcpy( dev_Iearr, h_Iearr, num_e*sizeof(int), hipMemcpyHostToDevice ) ; 
    hipMemcpy( dev_Icarr, h_Icarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_Isarr, h_Isarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipDeviceSynchronize();

/****************************Seed Path find, copy to device
******************************************************************************************************/    

    //find intial sead path
    vector<pair<int, pair<int,int>>> seedPath;
    seedPath = seedPathDijkstra( 0, 10, h_offset, h_offend,h_earr, h_carr, h_sarr, num_v, num_e);
    
    int pathSize = seedPath.size();
    int h_seedarr[3][pathSize];    
    
    cout<<"seedPath---";
    int iseed=0;
    for(auto v : seedPath)
    {
        cout<<"("<<v.first<<", "<<"("<<v.second.first<<","<<v.second.second<<")"<<endl;
        h_seedarr[0][iseed] = v.first;
        h_seedarr[1][iseed] = v.second.first;
        h_seedarr[2][iseed] = v.second.second;
        iseed++;
    }
 
 //alloate memory to device and copy
    int *d_seedarr;
    hipMalloc((void **)&d_seedarr,3*pathSize*sizeof(int));
    hipMemcpy( d_seedarr, h_seedarr, 3*pathSize*sizeof(int), hipMemcpyHostToDevice ) ;
    hipDeviceSynchronize();

/******************************************************************************************************/
 //Kernel call to check functionalities
  display<<<1,1>>>( pathSize, d_seedarr, dev_offset, dev_offend,dev_earr, dev_carr, dev_sarr, num_v, num_e, dev_Ioffset, dev_Ioffend,dev_Iearr, dev_Icarr, dev_Isarr );
    hipDeviceSynchronize();
/*****************************************************************************************************/ 
    hipFree(dev_offset);
    hipFree(dev_offend); 
    hipFree(dev_earr);
    hipFree(dev_carr);
    hipFree(dev_sarr);
    hipFree(d_seedarr); 
    hipFree(dev_Ioffset);
    hipFree(dev_Ioffend); 
    hipFree(dev_Iearr);
    hipFree(dev_Icarr);
    hipFree(dev_Isarr);
    hipDeviceSynchronize();
    return 0;

}