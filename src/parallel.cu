#include "hip/hip_runtime.h"
%%cu
#include<bits/stdc++.h>
using namespace std;

# define INF 0x3f3f3f3f

void printGrpah( int *a, int *e, int *b, int *c, int *d, int num_v, int num_e ) 
{

    for(int v = 0; v<num_v; v++)
    {
        if(a[v] != -1)
        {
            int start = a[v];
            int end = e[v];

            printf("%d-->", v);
            while( start <= end )
            {
                printf("[%d, {%d, %d} ]---", b[start], c[start], d[start]);
                start++;
            }
        }
        printf("\n");
    }
    return;
             
}

/************************ Dijkstras Algorithm for Intitial Seed Path Start ***********************************/
  
vector<pair<int, pair<int,int>>> seedPathDijkstra(int src, int dest, int* h_offset, int* h_offend, int* h_earr, int* h_carr, int* h_sarr, int num_v, int num_e) 
{ 
	priority_queue< pair<int, int>, vector <pair<int, int>> , greater <pair<int, int>> > pq;
    int V = num_v;
	vector<int> dist(V, INF);
    vector<pair<int, pair<int,int>>> parent(V, {-1, {-1,-1}});
    vector<pair<int, pair<int,int>>> seedPath;

	pq.push(make_pair(0, src));
	dist[src] = 0;
	
	vector<bool> f(V, false);

	while (!pq.empty())
	{
		int u = pq.top().second;
		pq.pop();
		f[u] = true;

        int index = h_offset[u];
        int end = h_offend[u];

		for (int i=index; i <=end; i++)
		{
            int v = h_earr[i];
            int cost = h_carr[i];
            int score = h_sarr[i];

			// If there is shorted path to v through u.
			if (f[v] == false && dist[v] > dist[u] + cost)
			{
				// Updating distance of v
				dist[v] = dist[u] + cost;
				pq.push(make_pair(dist[v], v));
                pair<int, pair<int,int>> temp;
                temp.first = u;
                temp.second.first = cost;
                temp.second.second = score;
                parent[v] = temp;
			}
		}
	}

    //Find shortest path from src to some dest
    stack<pair<int, pair<int,int>>> path;
    int nd = dest;
    while(parent[nd].first != -1)
    {
        path.push(parent[nd]);
        nd = parent[nd].first;
    }

    while(!path.empty())
    {
        seedPath.push_back(path.top());
        path.pop();
    }
    seedPath.push_back({dest, {0,0}});

    int sz_p = seedPath.size();
    for(int i=sz_p-1; i>0; i--)
    {
        seedPath[i].second.first = seedPath[i-1].second.first;
        seedPath[i].second.second = seedPath[i-1].second.second;        
    }
    seedPath[0].second.first = 0;
    seedPath[0].second.second = 0;

    for(int i=1; i<sz_p; i++)
    {
        seedPath[i].second.first += seedPath[i-1].second.first;
        seedPath[i].second.second += seedPath[i-1].second.second;        
    }
        

    return seedPath;      
} 
/********************* Dijkstra seed PAth end*******************************************/

/**************************** Kernel test code*********/

__global__ void display(int pathSize, int* d_seedarr,int* dev_offset, int* dev_offend,int* dev_earr,int* dev_carr,int* dev_sarr,int num_v,int num_e) 
{
    return;
}
/****************************************************/

int main()
{

/****************************Graph Initialisation in host and device
*************************************************************************************/    
    int num_v = 11;
    int num_e = 10;   

    int h_offset[num_v] = {0,1,4,5,6,7,-1,8,9,-1,-1};
    int h_offend[num_v] = {0,3,4,5,6,7,-1,8,9,-1,-1};     
    int h_earr[num_e] = {1,2,7,3,5,4,5,10,8,5};
    int h_carr[num_e] = {2,5,6,16,7,17,4,3,9,3}; 
    int h_sarr[num_e] = {1,3,4,10,0,15,2,5,15,0};    //variables declared for representing graph in cpu
    int *dev_offset, *dev_offend, *dev_earr, *dev_carr, *dev_sarr;  

    hipMalloc( (void**)&dev_offset, num_v*sizeof(int) ) ;
    hipMalloc( (void**)&dev_offend, num_v*sizeof(int) ) ;    
    hipMalloc( (void**)&dev_earr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_carr, num_e*sizeof(int) ) ;
    hipMalloc( (void**)&dev_sarr, num_e*sizeof(int) ) ; 

    hipMemcpy( dev_offset, h_offset, num_v*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_offend, h_offend, num_v*sizeof(int), hipMemcpyHostToDevice ) ;    
    hipMemcpy( dev_earr, h_earr, num_e*sizeof(int), hipMemcpyHostToDevice ) ; 
    hipMemcpy( dev_carr, h_carr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;
    hipMemcpy( dev_sarr, h_sarr, num_e*sizeof(int), hipMemcpyHostToDevice ) ;

    printGrpah(h_offset, h_offend,h_earr, h_carr, h_sarr, num_v, num_e);

/****************************Seed Path find, copy to device
*************************************************************************************/    

    //find intial sead path
    vector<pair<int, pair<int,int>>> seedPath;
    seedPath = seedPathDijkstra( 0, 10, h_offset, h_offend,h_earr, h_carr, h_sarr, num_v, num_e);
    
    int pathSize = seedPath.size();
    int h_seedarr[3][pathSize];    
    
    cout<<"seedPath---";
    int iseed=0;
    for(auto v : seedPath)
    {
        cout<<"("<<v.first<<", "<<"("<<v.second.first<<","<<v.second.second<<")"<<endl;
        h_seedarr[0][iseed] = v.first;
        h_seedarr[1][iseed] = v.second.first;
        h_seedarr[2][iseed] = v.second.second;
        iseed++;
    }
 
 //alloate memory to device and copy
    int *d_seedarr;
    hipMalloc((void **)&d_seedarr,3*pathSize*sizeof(int));
    hipMemcpy( d_seedarr, h_seedarr, 3*pathSize*sizeof(int), hipMemcpyHostToDevice ) ;


/******************************************************************************************************/
 //Kernel call to display global device memory
   display<<<1,1>>>( pathSize, d_seedarr, dev_offset, dev_offend,dev_earr, dev_carr, dev_sarr, num_v, num_e );


/*****************************************************************************************************/ 
    hipFree(dev_offset);
    hipFree(dev_offend); 
    hipFree(dev_earr);
    hipFree(dev_carr);
    hipFree(dev_sarr);
    hipFree(d_seedarr); 
 
    return 0;

}